#include "hip/hip_runtime.h"
#include "include/fast-lopq/model.cuh"

#include <fstream>
#include <iostream>
#include <limits>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "lopq_model.pb.h"


#define IDX(i, j, s) (((j)*(s))+(i))
#define FINITIALIZER (std::numeric_limits<scalar_t>::infinity())


namespace {

template <class T>
void log1d(const std::string& name, T* x, int w) {
	T* x_ = new T[w];
	hipMemcpy(x_, x, w * sizeof(T), hipMemcpyDeviceToHost);

	std::cout << name << "\n";
	for (int i = 0; i < w; ++i) {
		std::cout << x_[i] << ", ";
		if ((i+1) % 5 == 0)
			std::cout << '\n';
	}
	std::cout << '\n';

	delete[] x_;
}

void log2d(const std::string& name, scalar_t* x_, int d) {
	scalar_t* x = new scalar_t[256*64];
	hipMemcpy(x, x_, 256*64 * sizeof(scalar_t), hipMemcpyDeviceToHost);

//	std::cout << name << "\n";
//	for (int i = 0; i < 4; ++i) {
//		for (int j = 0; j < 4; ++j)
//			std::cout << x[IDX(i, j, d)] << ", ";
//		std::cout << '\n';
//	}
//	std::cout << '\n';


	std::cout << name << " samples\n";
	std::cout << 64 << "-" << 0 << ": " << x[IDX(64, 0, d)] << "\n";
	std::cout << 64 << "-" << 1 << ": " << x[IDX(64, 1, d)] << "\n";

	delete x;
}

__global__
void susq(const scalar_t* x_, const scalar_t* C_, uint8_t cszw, scalar_t* ds_) {
	for (int i = 0; i < cszw; ++i) {
		auto v = static_cast<scalar_t>(x_[i] - C_[IDX(threadIdx.x, i, blockDim.x)]);

		ds_[threadIdx.x] += v * v;
	}
}

__global__
void residual(scalar_t* r_, const scalar_t* x_, uint8_t sz, const uint8_t cluster, const scalar_t* C_, const int csz, const scalar_t* mu_) {
	for (int i = 0; i < sz; ++i)
		r_[i] = static_cast<scalar_t>(x_[i] - C_[IDX(cluster, i, csz)] - mu_[i]);
}

} // namespace


namespace lopq {
namespace gpu {

Model::Model(hipblasHandle_t handle) : handle(handle) {

}

void Model::load(const std::string& proto_path) {
	com::flickr::vision::lopq::LOPQModelParams lopq_params;

	std::ifstream proto_stream(proto_path);
	lopq_params.ParseFromIstream(&proto_stream);

	num_coarse_splits = lopq_params.cs_size();
	num_fine_splits = lopq_params.subs_size() / 2;

	assert(num_coarse_splits);
	assert(num_fine_splits);

	//TODO Check for cuda/cublas response statuses

	Cs = new scalar_t*[num_coarse_splits];
	Cszs = new Size[num_coarse_splits];
	for (uint32_t ci = 0; ci < num_coarse_splits; ++ci) {
		const auto& cs = lopq_params.cs(ci);
		Cszs[ci].h = cs.shape(0);
		Cszs[ci].w = cs.shape(1);
		auto& sz = Cszs[ci];

		std::cout << "Cs[" << ci << "]: " << sz.h << "x" << sz.w << '\n';

		scalar_t C[sz.w * sz.h];
		for (uint32_t i = 0; i < sz.h; ++i)
			for (uint32_t j = 0; j < sz.w; ++j)
				C[IDX(i, j, sz.h)] = cs.values(i * sz.w + j);

		hipMalloc((void**)&Cs[ci], sz.w * sz.h * sizeof(scalar_t));
		hipMemset(Cs[ci], FINITIALIZER, sz.h * sz.w * sizeof(scalar_t));
		hipMemcpy(Cs[ci], C, sz.h * sz.w * sizeof(scalar_t), hipMemcpyHostToDevice);
	}

	Rs = new scalar_t**[2];
	uint32_t rs_size = lopq_params.rs_size();
	uint32_t rs_half = rs_size / 2;
	for (uint32_t ri = 0; ri < 2; ++ri)
		Rs[ri] = new scalar_t*[rs_half];
	for (uint32_t c = 0; c < rs_size; ++c) {
		const auto& rs = lopq_params.rs(c);

		Size sz;
		sz.h = rs.shape(0);
		sz.w = rs.shape(1);
		if (c % rs_half == 0)
			std::cout << '\n';
		std::cout << "\rRs[" << c / rs_half << ", " << c % rs_half << "]: " << sz.h << "x" << sz.w;

		scalar_t R[sz.w * sz.h];
		for (uint32_t i = 0; i < sz.h; ++i) {
			for (uint32_t j = 0; j < sz.w; ++j)
				R[IDX(i, j, sz.h)] = rs.values(i * sz.w + j);
		}

		auto& R_ = Rs[c / rs_half][c % rs_half];
		hipMalloc((void**)&R_, sz.w * sz.h * sizeof(*R_));
		hipblasSetMatrix(sz.w, sz.h, sizeof(scalar_t), R, sz.w, R_, sz.w);
	}
	std::cout << '\n';

	mus = new scalar_t**[2];
	uint32_t mus_size = lopq_params.mus_size();
	uint32_t mus_half = mus_size / 2;
	for (uint32_t mui = 0; mui < 2; ++mui)
		mus[mui] = new scalar_t*[mus_half];
	for (uint32_t c = 0; c < mus_size; ++c) {
		const auto& mu = lopq_params.mus(c);
		auto sz = mu.values_size();
		if (c % mus_half == 0)
			std::cout << '\n';
		std::cout << "\rmu[" << c / mus_half << ", " << c % mus_half << "]: " << sz;

		scalar_t muc[sz];
		for (uint32_t i = 0; i < sz; ++i)
			muc[i] = mu.values(i);

		auto& mu_ = mus[c / mus_half][c % mus_half];
		hipMalloc((void**)&mu_, sz * sizeof(scalar_t));
		hipMemset(mu_, FINITIALIZER, sz * sizeof(scalar_t));
		hipblasSetVector(sz, sizeof(scalar_t), muc, 1, mu_, 1);
		// hipMemcpy(mu_, muc, sz * sizeof(scalar_t), hipMemcpyHostToDevice);

	}
	std::cout << '\n';

	subquantizers = new scalar_t**[2];
	uint32_t subs_size = lopq_params.subs_size();
	uint32_t subs_half = subs_size / 2;
	for (uint32_t si = 0; si < 2; ++si)
		subquantizers[si] = new scalar_t*[subs_half];
	for (uint32_t c = 0; c < subs_size; ++c) {
		const auto& subs = lopq_params.subs(c);

		Size sz;
		sz.h = subs.shape(0);
		sz.w = subs.shape(1);
		if (c % subs_half == 0)
			std::cout << '\n';
		std::cout << "\rsubquantizers[" << c / subs_half << ", " << c % subs_half << "]: " << sz.h << "x" << sz.w;

		scalar_t S[sz.w * sz.h];
		for (uint32_t i = 0; i < sz.h; ++i) {
			for (uint32_t j = 0; j < sz.w; ++j)
				S[IDX(i, j, sz.h)] = subs.values(i * sz.w + j);
		}

		auto& S_ = subquantizers[c / subs_half][c % subs_half];
		hipMalloc((void**)&S_, sz.w * sz.h * sizeof(*S_));
		hipblasSetMatrix(sz.w, sz.h, sizeof(scalar_t), S, sz.w, S_, sz.w);
	}
	std::cout << '\n';
}

Model::Codes Model::predict_coarse(const scalar_t* x, const uint32_t sz) const {
	scalar_t* x_;
	hipMalloc((void**)&x_, sz * sizeof(scalar_t));
	hipblasSetVector(sz, sizeof(scalar_t), x, 1, x_, 1);

	Model::Codes coarse(num_coarse_splits);

	uint32_t split_size = sz / num_coarse_splits;
	for (uint32_t split = 0; split < num_coarse_splits; ++split)
		coarse.codes[split] = predict_cluster(&x_[split * split_size], split_size, Cs[split], Cszs[split].h);

	return coarse;
}

Model::Codes Model::predict_fine(const scalar_t* x, const uint32_t sz, const Model::Codes& coarse_code) const {
	scalar_t* x_;
	hipMalloc((void**)&x_, sz * sizeof(scalar_t));
	hipblasSetVector(sz, sizeof(scalar_t), x, 1, x_, 1);

	Model::Codes fine(num_fine_splits);

	auto px_ = project(x_, sz, coarse_code);

	uint32_t split_size = sz / num_coarse_splits;
	for (uint32_t split = 0; split < num_coarse_splits; ++split) {
		// Compute subquantizer codes
		uint32_t subsplit_size = split_size / num_fine_splits;
		for (uint32_t subsplit = 0; subsplit < num_fine_splits; ++subsplit) {
			fine.codes[split * num_fine_splits + subsplit] = predict_cluster(&px_.x[split * split_size + subsplit * subsplit_size], subsplit_size, subquantizers[split][subsplit], Cszs[split].h);
		}
	}

	return fine;
}

Model::Vector_<scalar_t> Model::project(const scalar_t* x_, const uint32_t sz, const Model::Codes& coarse_code) const {
	auto px_ = Vector_<scalar_t>(sz);

	uint32_t split_size = sz / num_coarse_splits;

	scalar_t* r_;
	hipMalloc((void**)&r_, sz * sizeof(r_[0]));
	hipMemset(r_, 0.0, sz * sizeof(r_[0]));

	for (uint32_t split = 0; split < num_coarse_splits; ++split) {
		auto& cluster = coarse_code.codes[split];

		residual<<<1, 1>>>(&r_[split * split_size], &x_[split * split_size], split_size, cluster, Cs[split], Cszs[split].h, mus[split][cluster]);

		const scalar_t alfa=1.0;
		const scalar_t beta=0;
		cublasgemv(handle, HIPBLAS_OP_N, split_size, split_size, &alfa, Rs[split][cluster], split_size, &r_[split * split_size], 1, &beta, &px_.x[split * split_size], 1);
	}

	hipFree(r_);

	return px_;
}

uint8_t Model::predict_cluster(scalar_t* x, const uint32_t sz, scalar_t* centroids, const uint32_t csz) const {
	scalar_t* ds_;
	hipMalloc((void**)&ds_, csz * sizeof(ds_[0]));
	hipMemset(ds_, 0.0, csz * sizeof(ds_[0]));
	susq<<<1, csz>>>(x, centroids, sz, ds_);
	hipDeviceSynchronize();

	int amin;
	hipblasIsamin(handle, csz, ds_, 1, &amin);

	hipFree(ds_);

	return (uint8_t)(amin - 1);
}

} // gpu
} // lopq
