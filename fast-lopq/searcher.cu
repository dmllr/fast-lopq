#include "hip/hip_runtime.h"
#include "include/fast-lopq/searcher.cuh"

#include <cmath>
#include <numeric>
#include <fstream>
#include <algorithm>
#include <memory>
#include <cassert>

#include <iostream>

#define BLOCK_SIZE 32

namespace lopq {
namespace gpu {

__device__
scalar_t distance(const lopq::gpu::Model& model, const scalar_t* x_, const size_t sz, const lopq::gpu::Model::Codes& coarse_code, const uint8_t* fine_code_) {
	scalar_t D = 99.55;

	printf("3. model.num_fine_splits, model.num_clusters: %d, %d\n", model.num_fine_splits, model.num_clusters);

	printf("d0\n");
	// scalar_t* d0_ = malloc(model.num_fine_splits * model.num_clusters * sizeof(scalar_t));
	// auto d0_ = new scalar_t[model.num_fine_splits * model.num_clusters];
	printf("%d, %d\n", model.num_fine_splits, model.num_clusters);
	scalar_t d0_[8 * 256];
	printf("malloc yes\n");
	model.subquantizer_distances_dododo(d0_, x_, sz, coarse_code.x, 0);
	printf("subq yes\n");
	auto d0s = 128;  // TODO replace 128
	printf("d0 yes\n");

	// printf("d1\n");
	// scalar_t* d1_ = (scalar_t*)malloc(model.num_fine_splits * model.num_clusters * sizeof(scalar_t));
	// lopq::gpu::subquantizer_distances(model, d1_, x_, sz, coarse_code.x, 1);
	// printf("d1 yes\n");

	// for (uint32_t i = 0; i < model.num_fine_splits; ++i) {
	// 	auto& e = fine_code_[i];
	// 	D += (i < d0s) ? d0_[i * model.num_clusters + e] : d1_[(i - d0s) * model.num_clusters + e];
	// };
	// printf("loop yes\n");
	
	// free(d0_);
	// free(d1_);
	printf("free yes\n");

	return D;
}

__global__
void all_distances(const lopq::gpu::Model& model, const scalar_t* x_, const size_t sz, const lopq::gpu::Model::Codes& coarse_code, const int n, const uint8_t* vectors_, scalar_t* distances_) {
	printf("2. model.num_fine_splits, model.num_clusters: %d, %d\n", model.num_fine_splits, model.num_clusters);

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = idx; i < n; i += stride) {
		auto fine_code_ = &vectors_[i * 16];  // TODO replace 16
		distances_[i] = distance(model, x_, sz, coarse_code, fine_code_);
	}
}

Searcher::Searcher(hipblasHandle_t handle) : handle(handle) {
	Model m(handle);
	model = m;
}

void Searcher::load_model(const std::string& proto_path) {
	model.load(proto_path);
}

scalar_t Searcher::distance(const scalar_t* x_, const size_t sz, const Model::Codes& coarse_code, const Model::Codes& fine_code, Searcher::DistanceCache& cache) const {
	scalar_t D = 0.0;

	auto& d0 = cache[coarse_code[0]];
	auto& d1 = cache[coarse_code[1]];
	auto d0s = d0.size;

	if (d0s == 0) {
		d0 = model.subquantizer_distances(x_, sz, coarse_code, 0);
		d0s = d0.size;
	}
	if (d1.size == 0)
		d1 = model.subquantizer_distances(x_, sz, coarse_code, 1);

	for (uint32_t i = 0; i < model.num_fine_splits; ++i) {
		auto& e = fine_code[i];
		D += (i < d0s) ? d0[i][e] : d1[i - d0s][e];
	};

	return D;
}

std::vector<Searcher::Response> Searcher::search(const scalar_t* x_) {
	auto coarse_code = model.predict_coarse(x_, 128);

	return search_in(coarse_code, x_, 128);
}

std::vector<Searcher::Response> Searcher::search_in(const Model::Codes& coarse_code, const scalar_t* x_, const size_t sz) {
	auto& index = get_cell(coarse_code);

	auto cluster_size = index.ids.size();
	const auto& index_codes_ = index.vectors;

	if (cluster_size == 0)
		return std::vector<Response>();
	
	// Searcher::DistanceCache distance_cache;

	using i_d = std::pair<uint, float>;

	std::vector<i_d> distances(cluster_size);

	// std::cout << "cluster_size: " << cluster_size << "\n";
	printf("1. model.num_fine_splits, model.num_clusters: %d, %d\n", model.num_fine_splits, model.num_clusters);

	// calculate relative distances for all vectors in cluster
	scalar_t* distances_;
	hipMalloc((void**)&distances_, cluster_size * sizeof(scalar_t));
	hipMemset(distances_, 0, cluster_size * sizeof(scalar_t));

	// all_distances<<<(cluster_size / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(model, x_, sz, coarse_code, cluster_size, index_codes_, distances_);
	all_distances<<<1, 1>>>(model, x_, sz, coarse_code, cluster_size, index_codes_, &distances_[0]);

	auto ldistances = new scalar_t[cluster_size];
	for (int i = 0; i < 4; ++i)
		ldistances[i] = 11.99;

	hipMemcpy(ldistances, distances_, cluster_size * sizeof(scalar_t), hipMemcpyDeviceToHost);

	// std::cout << "ldistances: ";
	// for (int i = 0; i < 4; ++i)
	// 	std::cout << ldistances[i] << " ";
	// std::cout << "\n";

	uint32_t c = 0;
	for (auto& e: distances) {
		e.second = ldistances[c];
		e.first = c++;
	};
	
	delete[] ldistances;

	 // take top N
	std::partial_sort(
			distances.begin(), distances.begin() + 12, distances.end(),
			[](i_d i1, i_d i2) {
				return i1.second < i2.second;
			}
	);

	std::vector<Searcher::Response> top;

	top.reserve(12);

	for(int i = 0; i < 12; ++i) {
		top.emplace_back(Response(index.ids[distances[i].first]));
	}

	return top;
}

} // gpu
} // lopq
